#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>
#include "timer.h"

using namespace std;

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}


__global__ void histogramKernel(unsigned char* image, long img_size, unsigned int* histogram, int hist_size) {
// insert operation here

}

void histogramCuda(unsigned char* image, long img_size, unsigned int* histogram, int hist_size) {
    int threadBlockSize = 512;

    // allocate the vectors on the GPU
    unsigned char* deviceImage = NULL;
    checkCudaCall(hipMalloc((void **) &deviceImage, img_size * sizeof(unsigned char)));
    if (deviceImage == NULL) {
        cout << "could not allocate memory!" << endl;
        return;
    }
    unsigned int* deviceHisto = NULL;
    checkCudaCall(hipMalloc((void **) &deviceHisto, hist_size * sizeof(unsigned int)));
    if (deviceHisto == NULL) {
        checkCudaCall(hipFree(deviceImage));
        cout << "could not allocate memory!" << endl;
        return;
    }

    timer kernelTime1 = timer("kernelTime1");
    timer memoryTime = timer("memoryTime");

    // copy the original vectors to the GPU
    memoryTime.start();
    checkCudaCall(hipMemcpy(deviceImage, image, img_size*sizeof(unsigned char), hipMemcpyHostToDevice));
    memoryTime.stop();

    // execute kernel
    kernelTime1.start();
    histogramKernel<<<img_size/threadBlockSize, threadBlockSize>>>(deviceImage, img_size, deviceHisto, hist_size);
    hipDeviceSynchronize();
    kernelTime1.stop();

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    memoryTime.start();
    checkCudaCall(hipMemcpy(histogram, deviceHisto, hist_size * sizeof(unsigned int), hipMemcpyDeviceToHost));
    memoryTime.stop();

    checkCudaCall(hipFree(deviceImage));
    checkCudaCall(hipFree(deviceHisto));

    cout << "histogram (kernel): \t\t" << kernelTime1  << endl;
    cout << "histogram (memory): \t\t" << memoryTime << endl;
}

void histogramSeq(unsigned char* image, long img_size, unsigned int* histogram, int hist_size) {
  int i; 

  timer sequentialTime = timer("Sequential");
  
  for (i=0; i<hist_size; i++) histogram[i]=0;

  sequentialTime.start();
  for (i=0; i<img_size; i++) {
	histogram[image[i]]++;
  }
  sequentialTime.stop();
  
  cout << "histogram (sequential): \t\t" << sequentialTime << endl;

}

int main(int argc, char* argv[]) {
    long img_size = 655360;
    int hist_size = 256;
    
    if (argc > 1) img_size = atoi(argv[1]);
    if (img_size < 1024) {
	cout << "Error in parameter" << endl;
	exit(-1);
    }

    unsigned char *image = (unsigned char *)malloc(img_size * sizeof(unsigned char)); 
    unsigned int *histogramS = (unsigned int *)malloc(hist_size * sizeof(unsigned int));     
    unsigned int *histogram = (unsigned int *)malloc(hist_size * sizeof(unsigned int));

    // initialize the vectors.
    for(long i=0; i<img_size; i++) {
        image[i] = (unsigned char) (i % hist_size);
    }

    cout << "Compute the histogram of a gray image with " << img_size << " pixels." << endl;

    histogramSeq(image, img_size, histogramS, hist_size);
    histogramCuda(image, img_size, histogram, hist_size);
    
    // verify the resuls
    for(int i=0; i<hist_size; i++) {
	  if (histogram[i]!=histogramS[i]) {
            cout << "error in results! Bin " << i << " is "<< histogram[i] << ", but should be " << histogramS[i] << endl; 
            exit(1);
        }
    }
    cout << "results OK!" << endl;
     
    free(image);
    free(histogram);
    free(histogramS);         
    
    return 0;
}
