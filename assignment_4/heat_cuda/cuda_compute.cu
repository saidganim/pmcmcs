#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <input.h>
#include <output.h>

#define _index_macro(a, b, c) a[M * (b) + (c)]
#define _index_macro_pat(a, b, c) a[(M + 2) * (b) + (c)]

static void checkCudaCall(hipError_t result) {
        if (result != hipSuccess) {
                printf("cuda error \n");
                exit(1);
        }
}

__device__ static double atomicMaxf(double* address, double val)
{
        unsigned long long int* address_as_i = (unsigned long long int*) address;
        unsigned long long int old = *address_as_i, assumed;
        do {
                assumed = old;
                old = atomicCAS(address_as_i, assumed,
                                __double_as_longlong(fmaxf(val,__longlong_as_double(assumed))));
        } while (assumed != old);
        return __longlong_as_double(old);
}


__global__ void iteration(double* temp_init, double* temp_tmp, double* conductivity, int N, int M, double* maxdiff) {
        unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned j = i % M + 1;
        double dir_nc = sqrtf(2)/(sqrtf(2) + 1) / 4;
        double dig_nc = 1 /(sqrtf(2) + 1) / 4;
        double diff;
        i = i / M + 1;
        if(i > N)
                return;

        *maxdiff = 0;
        for(int i = 0; i < N; ++i) {
                _index_macro_pat(temp_init, i + 1, 0) = _index_macro_pat(temp_init, i + 1, M); // move last column to 0's
                _index_macro_pat(temp_init, i + 1,M + 1) = _index_macro_pat(temp_init, i + 1, 1); // move first column to (M+1)'s
                _index_macro_pat(temp_tmp, i + 1,0) = _index_macro_pat(temp_init, i + 1, M); // move last column to 0's
                _index_macro_pat(temp_tmp, i + 1, M + 1) = _index_macro_pat(temp_init, i + 1, 1); // move first column to (M+1)'s
        }
        double weighted_neighb = dir_nc *
                                 ( // Direct neighbors
                _index_macro_pat(temp_init, i - 1, j) + _index_macro_pat(temp_init, i, j - 1) +
                _index_macro_pat(temp_init, i + 1, j) + _index_macro_pat(temp_init, i, j + 1)
                                 ) + dig_nc *
                                 ( // Diagonal neighbors
                _index_macro_pat(temp_init, i - 1, j - 1) +_index_macro_pat(temp_init, i + 1, j - 1) +
                _index_macro_pat(temp_init, i - 1, j + 1) + _index_macro_pat(temp_init, i + 1, j + 1)
                                 );
        weighted_neighb *= (1 - _index_macro(conductivity, i - 1, j - 1));
        _index_macro_pat(temp_tmp, i, j) = _index_macro_pat(temp_init, i, j) * _index_macro(conductivity, i - 1, j - 1);
        _index_macro_pat(temp_tmp, i, j) += weighted_neighb;

        diff = fabs(_index_macro_pat(temp_tmp, i, j) - _index_macro_pat(temp_init, i, j));
        atomicMaxf(maxdiff, diff);
}
extern "C"
void cuda_do_compute(const struct parameters* p, struct results *r) {
        struct timeval before, after;
        int threadBlockSize = 512;
        double (*temp_init)[p->N + 2][p->M + 2] = (double (*)[p->N + 2][p->M + 2])malloc((p->N + 2) * (p->M + 2) * sizeof(double));
        double maxdiff = p->threshold + 1.;
        unsigned int iter = 0;
        double* deviceMaxdiff;
        double* deviceA;
        double* deviceB;
        double* deviceConductivity;
        double local_sum = 0;
        int N = p->N, M = p->M;
        hipMalloc((void**)&deviceMaxdiff, sizeof(uint64_t));
        hipMemset(deviceMaxdiff, 0, sizeof(double));

        checkCudaCall(hipMalloc((void **) &deviceA, ((p->N + 2) * (p->M + 2) * sizeof(double) )));
        checkCudaCall(hipMalloc((void **) &deviceB, ((p->N + 2) * (p->M + 2) * sizeof(double) )));
        checkCudaCall(hipMalloc((void **) &deviceConductivity, ((p->N) * (p->M) * sizeof(double) )));

        for(int i = 0; i < N; ++i)
                memcpy(&(*temp_init)[i + 1][1], &_index_macro(p->tinit, i, 0), M * sizeof(double));

        for(int i = 0; i < M; ++i) {
                (*temp_init)[0][i + 1] = (*temp_init)[1][i + 1];
                (*temp_init)[N + 1][i + 1] = (*temp_init)[N][i + 1];
        }
// Filling [0][0], [0][M + 1], [N + 1][0], [N + 1][M + 1] elems
        (*temp_init)[0][0] = (*temp_init)[0][M];
        (*temp_init)[0][M + 1] = (*temp_init)[0][1];
        (*temp_init)[N + 1][0] = (*temp_init)[N][M];
        (*temp_init)[N + 1][M + 1] = (*temp_init)[N + 1][1];


// copy the original vectors to the GPU
        checkCudaCall(hipMemcpy(deviceA, temp_init, (p->N + 2) * (p->M + 2) * sizeof(double), hipMemcpyHostToDevice));
        checkCudaCall(hipMemcpy(deviceB, temp_init, (p->N + 2) * (p->M + 2) * sizeof(double), hipMemcpyHostToDevice));
        checkCudaCall(hipMemcpy(deviceConductivity, p->conductivity, (p->N) * (p->M) * sizeof(double), hipMemcpyHostToDevice));


        gettimeofday(&before, NULL);

        while(iter++ < (p->maxiter) && maxdiff > p->threshold) {
// execute kernel
                maxdiff = 0;
                {double* tmp = deviceA; deviceA = deviceB; deviceB = tmp;}
                dim3 gridSize(N/threadBlockSize + 1, M / threadBlockSize + 1);
                iteration<<<(N * M / threadBlockSize + 1), threadBlockSize>>>(deviceA, deviceB, deviceConductivity, N, M, deviceMaxdiff);
                hipMemcpy(&maxdiff, deviceMaxdiff, sizeof(uint64_t), hipMemcpyDeviceToHost);
                if((iter % p->period) == 0){
            			local_sum = 0;
            			gettimeofday(&end, 0);
            			r->tmin = r->tmax = (*temp_tmp)[1][1];
            			for(int i = 1; i <= N; ++i){
            				for(int j = 1; j <= M ; ++j){
            					if((*temp_init)[i][j] > r->tmax)
            						r->tmax = (*temp_init)[i][j];
            					if((*temp_init)[i][j] < r->tmin)
            						r->tmin = (*temp_init)[i][j];
            					local_sum += (*temp_init)[i][j];
            				}
            			}
        }
        gettimeofday(&after, NULL);

// check whether the kernel invocation was successful
        checkCudaCall(hipGetLastError());

// copy result back
        checkCudaCall(hipMemcpy(temp_init, deviceB, (N + 2) * (M + 2) * sizeof(double), hipMemcpyDeviceToHost));
        r->tmin = r->tmax = (*temp_init)[1][1];
        local_sum = 0;
        for(int i = 1; i <= N; ++i) {
                for(int j = 1; j <= M; ++j) {
                        if((*temp_init)[i][j] > r->tmax)
                                r->tmax = (*temp_init)[i][j];
                        if((*temp_init)[i][j] < r->tmin)
                                r->tmin = (*temp_init)[i][j];
                        local_sum += (*temp_init)[i][j];
                }
        }
        r->niter = iter - 1;
        r->tavg = local_sum /(N * M);
        r->maxdiff = maxdiff;
        r->time = (double)(after.tv_sec - before.tv_sec) +
                  (double)(after.tv_usec - before.tv_usec) / 1e6;
        checkCudaCall(hipFree(deviceA));
        checkCudaCall(hipFree(deviceB));
        checkCudaCall(hipFree(deviceConductivity));
        checkCudaCall(hipFree(deviceMaxdiff));

}
